#include "hip/hip_runtime.h"
#include "convolution2d.cuh"

#include <opencv2/core/cuda/common.hpp>
#include <opencv2/cudev.hpp>
#include <hip/hip_runtime.h>
#include <>

__device__ uchar clipGpu(float val)
{
    return (val < 0.0f) ? 0 : (val > 255.0f) ? 255 : (uchar)val;
}

__global__ void convolution2dGpu
(
    const cv::cudev::PtrStepSz<uchar> src,
    cv::cudev::PtrStepSz<uchar> dst,
    const cv::cudev::PtrStepSz<float> kernel
)
{
    const int x = blockDim.x * blockIdx.x + threadIdx.x;
    const int y = blockDim.y * blockIdx.y + threadIdx.y;
    const int half_size = (kernel.cols / 2);

    if((y >= half_size) && y < (src.rows - half_size)){
        if((x >= half_size) && (x < (src.cols - half_size))){
            float sum = 0.0f;
            for (int dy = -half_size; dy <= half_size; dy++){
                for (int dx = -half_size; dx <= half_size; dx++){
                    sum = __fadd_rn(sum, __fmul_rn(kernel.ptr(dy+half_size)[dx+half_size], src.ptr(y+dy)[x+dx]));
                }
            }
            dst.ptr(y)[x] = clipGpu(sum);
        }
    }
}

void launch_convolution2dGpu
(
    const cv::cuda::GpuMat& src,
    cv::cuda::GpuMat& dst,
    const cv::cuda::GpuMat& kernel
)
{
    const dim3 block(64, 2);
    const dim3 grid(cv::cudev::divUp(dst.cols, block.x), cv::cudev::divUp(dst.rows, block.y));

    convolution2dGpu<<<grid, block>>>(src, dst, kernel);

    CV_CUDEV_SAFE_CALL(hipGetLastError());
    CV_CUDEV_SAFE_CALL(hipDeviceSynchronize());
}

double launch_convolution2dGpu
(
    const cv::cuda::GpuMat& src,
    cv::cuda::GpuMat& dst,
    const cv::cuda::GpuMat& kernel, 
    const int loop_num
)
{
    double f = 1000.0f / cv::getTickFrequency();
    int64 start = 0, end = 0;
    double time = 0.0;
    for (int i = 0; i <= loop_num; i++){
        start = cv::getTickCount();
        launch_convolution2dGpu(src, dst, kernel);
        end = cv::getTickCount();
        time += (i > 0) ? ((end - start) * f) : 0;
    }
    time /= loop_num;

    return time;
}
